#include "hip/hip_runtime.h"
#include "LBMSolver.h"

#include <GL/glew.h>
#include <GLFW/glfw3.h>

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// CUDA helper functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <queue>

#include "LBMKernels.cuh"
#include "colorMap.cuh"

#include <cmath>
#include <fstream>
#include <thrust/device_vector.h>


#define THREADS_PER_BLOCK 8

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

LBMSolver::LBMSolver( uint nx, uint ny, float omega, float U, float V )
    : nx(nx), ny(ny), omega(omega), U(U), V(V), lbModel('c'), geoMode('w')
{
	this->speed = sqrtf(U * U + V * V);
	this->alpha = acosf(U / sqrt((U*U + V * V + 1.e-20)));
    this->nx = nx;
    this->ny = ny;
    this->refLength=ny;

    this->f.f00 = std::make_shared<floatVec>( nx * ny );
    this->f.fp0 = std::make_shared<floatVec>( nx * ny );
    this->f.fn0 = std::make_shared<floatVec>( nx * ny );
    this->f.fpp = std::make_shared<floatVec>( nx * ny );
    this->f.fnp = std::make_shared<floatVec>( nx * ny );
    this->f.fpn = std::make_shared<floatVec>( nx * ny );
    this->f.fnn = std::make_shared<floatVec>( nx * ny );
    this->f.f0p = std::make_shared<floatVec>( nx * ny );
    this->f.f0n = std::make_shared<floatVec>( nx * ny );

    this->f.geo = std::make_shared<charVec> ( nx * ny );

    this->f.pressure = std::make_shared<floatVec>( nx * ny );
    this->f.velocity = std::make_shared<floatVec>( nx * ny );

    this->minPressure = -1.0e-3f;
    this->maxPressure =  1.0e-3f;

    this->minVelocity =  0.0f;
    this->maxVelocity =  0.02f;

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( colorMapDeviceR), colorMapHostR, 36*sizeof(float) ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( colorMapDeviceG), colorMapHostG, 36*sizeof(float) ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( colorMapDeviceB), colorMapHostB, 36*sizeof(float) ) );
}

LBMSolver::~LBMSolver()
{
}
void LBMSolver::connectVertexBuffer(uint vertexBufferID)
{
    hipError_t cudaStatus = hipGraphicsGLRegisterBuffer(&this->glVertexBufferResource, 
                                                          vertexBufferID, 
                                                          cudaGraphicsMapFlagsNone);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphicsGLRegisterBuffer failed: %s\n", hipGetErrorString(cudaStatus));
        // Additional error handling if necessary
    }
}

// void LBMSolver::connectVertexBuffer(uint vertexBufferID)
// {
//     // register the OpenGL vertex Buffer within CUDA
//     checkCudaErrors( hipGraphicsGLRegisterBuffer(&this->glVertexBufferResource, 
//                                                   vertexBufferID, 
//                                                   cudaGraphicsMapFlagsNone) );
    
// }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void LBMSolver::initializeDistributions()
{
    dim3 threads( THREADS_PER_BLOCK, THREADS_PER_BLOCK );
    dim3 blocks ( ( this->nx +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK,
                  ( this->ny +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK );

    initializeDistributionsKernel<<<blocks, threads>>>( this->getDistPtr(), this->nx, this->ny, this->U, this->V );

    swap( f.f0n, f.f0p );
    swap( f.fnn, f.fpp );
    swap( f.fp0, f.fn0 );
    swap( f.fpn, f.fnp );

    initializeDistributionsKernel<<<blocks, threads>>>( this->getDistPtr(), this->nx, this->ny, this->U, this->V );

    scaleColorMap();
}

void LBMSolver::initializeGeo()
{
    dim3 threads( THREADS_PER_BLOCK, THREADS_PER_BLOCK );
    dim3 blocks ( ( this->nx +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK,
                  ( this->ny +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK );

    initializeGeoKernel<<<blocks, threads>>>( this->getDistPtr(), this->nx, this->ny );

    scaleColorMap();
}

void LBMSolver::collision()
{
    dim3 threads( THREADS_PER_BLOCK, THREADS_PER_BLOCK );
    dim3 blocks ( ( this->nx +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK,
                  ( this->ny +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK );

    //////////////////////////////////////////////////////////////////////////

    collisionKernel<<<blocks, threads>>>( this->getDistPtr(), this->nx, this->ny, this->omega, this->U, this->V, this->lbModel );
    getLastCudaError("collisionKernel failed.");

    //////////////////////////////////////////////////////////////////////////

    swap( f.f0n, f.f0p );
    swap( f.fnn, f.fpp );
    swap( f.fp0, f.fn0 );
    swap( f.fpn, f.fnp );
}

void LBMSolver::postProcessing( char type )
{
    this->computeMacroscopicQuantities();

    dim3 threads( THREADS_PER_BLOCK, THREADS_PER_BLOCK );
    dim3 blocks ( ( this->nx +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK,
                  ( this->ny +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK );

    hipGraphicsMapResources(1, &this->glVertexBufferResource, 0);
    getLastCudaError("hipGraphicsMapResources failed");

    float* verticesDev;
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&verticesDev, &num_bytes, this->glVertexBufferResource);
    getLastCudaError("hipGraphicsResourceGetMappedPointer failed");

    float min;
    float max;

    if( type == 'p' ){
        min = this->minPressure;
        max = this->maxPressure;
    }
    else{
        min = this->minVelocity;
        max = this->maxVelocity;
    }

    postProcessingSetColorKernel<<<blocks, threads>>>( this->getDistPtr(), this->nx, this->ny, verticesDev, type, min, max, this->geoMode );
    getLastCudaError("postProcessingSetColorKernel failed.");

    hipGraphicsUnmapResources(1, &this->glVertexBufferResource, 0);
    getLastCudaError("hipGraphicsUnmapResources failed");
}

void LBMSolver::computeMacroscopicQuantities()
{
    dim3 threads( THREADS_PER_BLOCK, THREADS_PER_BLOCK );
    dim3 blocks ( ( this->nx +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK,
                  ( this->ny +  THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK );

    postProcessingMacroscopicQuantitiesKernel<<<blocks, threads>>>( this->getDistPtr(), this->nx, this->ny );
    getLastCudaError("postProcessingMacroscopicQuantitiesKernel failed.");
}

void LBMSolver::scaleColorMap()
{
    this->computeMacroscopicQuantities();

    this->minPressure = thrust::reduce( this->f.pressure->begin(), this->f.pressure->end(),  10.0f, thrust::minimum<float>() );
    this->maxPressure = thrust::reduce( this->f.pressure->begin(), this->f.pressure->end(), -10.0f, thrust::maximum<float>() );

    std::cout << "Pressure = ( " << this->minPressure << ", " << this->maxPressure << " )" << std::endl;

    this->minVelocity = thrust::reduce( this->f.velocity->begin(), this->f.velocity->end(),  10.0f, thrust::minimum<float>() );
    this->maxVelocity = thrust::reduce( this->f.velocity->begin(), this->f.velocity->end(), -10.0f, thrust::maximum<float>() );

    std::cout << "Velocity = ( " << this->minVelocity << ", " << this->maxVelocity << " )" << std::endl;
}

void LBMSolver::setGeo(uint xIdx, uint yIdx, char geo)
{
    dim3 threads ( 2, 2 );

    if( geo == GEO_FLUID ){
        threads.x += 4;
        threads.y += 4;
    }

    setGeoKernel<<<1, threads>>>( this->getDistPtr(), this->nx, this->ny, xIdx, yIdx, geo );
}

void LBMSolver::setGeo(uint xIdx1, uint yIdx1, uint xIdx2, uint yIdx2, char geo)
{
    int dxIdx = xIdx2 - xIdx1;
    int dyIdx = yIdx2 - yIdx1;

    if( abs(dxIdx) >= abs(dyIdx) ){
        for( uint idx = 0; idx < abs(dxIdx); idx++ ){
    
            float xInc = ( dxIdx != 0 )?( float(dxIdx) / float( abs(dxIdx) ) ):(0);
            float yInc = ( dxIdx != 0 )?( float(dyIdx) / float( abs(dxIdx) ) ):(0);
            
            // int x = int(xIdx1) + float(idx) * xInc;
            // int y = int(yIdx1) + float(idx) * yInc;
            float x = xIdx1 + static_cast<float>(idx) * xInc;
            float y = yIdx1 + static_cast<float>(idx) * yInc;

            this->setGeo(x,y, geo);
        }
    }else{
        for( uint idx = 0; idx < abs(dyIdx); idx++ ){
    
            float xInc = ( dyIdx != 0 )?( float(dxIdx) / float( abs(dyIdx) ) ):(0);
            float yInc = ( dyIdx != 0 )?( float(dyIdx) / float( abs(dyIdx) ) ):(0);

            // int x = int(xIdx1) + float(idx) * xInc;
            // int y = int(yIdx1) + float(idx) * yInc;
            float x = xIdx1 + static_cast<float>(idx) * xInc;
            float y = yIdx1 + static_cast<float>(idx) * yInc;

            this->setGeo(x,y, geo);
        }
    }
}

void LBMSolver::setGeoFloodFill(uint xIdx, uint yIdx, char geo)
{
    // based on
    // https://stackoverflow.com/questions/30608448/flood-fill-recursive-stack-overflow

    // download geo field
    charVecHost hostGeo = *this->f.geo;

    //setGeoFloodFillRecursion( xIdx, yIdx, geo, hostGeo );

    struct coordinate { uint x, y; };
    std::queue<coordinate> to_draw;
    to_draw.push({xIdx, yIdx});

    while (!to_draw.empty())
    {
        auto top = to_draw.front();
        to_draw.pop();

        if( top.x <= 0 || top.y <= 0 || top.x >= nx - 2 || top.y >= ny - 2 ) continue;

        uint nodeIdx = top.x + top.y * nx;

        if( hostGeo[ nodeIdx ] == geo ) continue;

        hostGeo[ nodeIdx ] = geo;

        to_draw.push( { top.x, top.y + 1 } );
        to_draw.push( { top.x, top.y - 1 } );
        to_draw.push( { top.x + 1, top.y } );
        to_draw.push( { top.x - 1, top.y } );
    }

    // upload geo field
    *this->f.geo = hostGeo;
}


void LBMSolver::setNu(float nu)
{
    if( nu < 1.0e-8f ) nu = 1.0e-8f;
    if( nu > 0.1f )     nu = 0.1f;
    this->omega = 1.0f / ( 3.0f * nu + 0.5f );
}

float LBMSolver::getNu()
{
    return ( 1.0f / this->omega - 0.5f ) / 3.0f;
}

void LBMSolver::setU(float U)
{
    if( U >  0.1f ) U =  0.1f;
    if( U < -0.1f ) U = -0.1f;
    this->U = U;
}

void LBMSolver::setV(float V)
{
    if( V >  0.1f ) V =  0.1f;
    if( V < -0.1f ) V = -0.1f;
    this->V = V;
}

void LBMSolver::setAlpha(float alpha)
{
	this->alpha = alpha;
}

void LBMSolver::setSpeed(float speed)
{
	this->speed = speed;
	setU(speed*cosf(alpha));
	setV(speed*sinf(alpha));
}

void LBMSolver::setRefLength(uint ref)
{
	this->refLength = ref;
}

float LBMSolver::getU()
{
    return this->U;
}

float LBMSolver::getV()
{
    return this->V;
}

float LBMSolver::getAlpha()
{
	return this->alpha;
}

float LBMSolver::getSpeed()
{
	return this->speed;
}

uint LBMSolver::getRefLength()
{
	return this->refLength;
}

void LBMSolver::setLBModel(char lbModel)
{
    this->lbModel = lbModel;
}

char LBMSolver::getLBModel()
{
    return this->lbModel;
}

void LBMSolver::setGeoMode(char geoMode)
{
    this->geoMode = geoMode;
}

char LBMSolver::getGeoMode()
{
    return this->geoMode;
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void LBMSolver::swap( floatVecPtr& lhs, floatVecPtr& rhs )
{
    floatVecPtr tmp = lhs;
    lhs = rhs;
    rhs = tmp;
}

uint LBMSolver::c2i( uint xIdx, uint yIdx )
{
    return yIdx * this->nx + xIdx;
}

D2Q9Ptr LBMSolver::getDistPtr()
{
    D2Q9Ptr distPtr;

    distPtr.f00 = this->f.f00->data();
    distPtr.fp0 = this->f.fp0->data();
    distPtr.fn0 = this->f.fn0->data();
    distPtr.fpp = this->f.fpp->data();
    distPtr.fnp = this->f.fnp->data();
    distPtr.fpn = this->f.fpn->data();
    distPtr.fnn = this->f.fnn->data();
    distPtr.f0p = this->f.f0p->data();
    distPtr.f0n = this->f.f0n->data();
    distPtr.geo = this->f.geo->data();

    distPtr.velocity = this->f.velocity->data();
    distPtr.pressure = this->f.pressure->data();

    return distPtr;
}

float LBMSolver::getVelocity(uint index)
{
    thrust::device_vector<float> velocity = *f.velocity;
    return velocity[index];
}

float LBMSolver::getPressure(uint index)
{
    thrust::device_vector<float> pressure = *f.pressure;
    return pressure[index];
}
 std::vector<float> LBMSolver::getVelocityData() { 
    thrust::device_vector<float>& velocityData = *f.velocity;
    std::vector<float> velocityDataHost(velocityData.begin(), velocityData.end());
    return velocityDataHost;
    
    
     }
     std::vector<float> LBMSolver::getPressureData() { 
    thrust::device_vector<float>& PressureData = *f.pressure;
    std::vector<float> PressureDataHost(PressureData.begin(), PressureData.end());
    return PressureDataHost;
    
    
     }
   

   std::vector<char> LBMSolver::getgeoData() {
        thrust::device_vector<char>& geoprofile = *f.geo;
        std::vector<char> geoprofileHost(geoprofile.size());
        thrust::copy(geoprofile.begin(), geoprofile.end(), geoprofileHost.begin());

        return geoprofileHost;
    }

void LBMSolver::setgeoData(const std::vector<char>& data) {
        thrust::device_vector<char>& geoprofile = *f.geo;
        geoprofile.resize(data.size());
        thrust::copy(data.begin(), data.end(), geoprofile.begin());
    }

 


