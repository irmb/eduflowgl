#include "hip/hip_runtime.h"
#include "cudaKernel.h"

hipGraphicsResource* glVertexBufferResource;

__global__ void postProcessingSetColorKernel( float* vertices, float delta )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    vertices[ 6 * idx + 0 ] = vertices[ 6 * idx + 0 ] + delta;
}

void changeTriangle( float delta )
{
    hipGraphicsMapResources(1, &glVertexBufferResource, 0);

    float* verticesDev;
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&verticesDev, &num_bytes, glVertexBufferResource);

    postProcessingSetColorKernel<<<1, 3>>>(verticesDev, delta);

    hipGraphicsUnmapResources(1, &glVertexBufferResource, 0);
}

void connectVertexBuffer(unsigned int vertexBufferID)
{
    // register the OpenGL vertex Buffer within CUDA
    hipGraphicsGLRegisterBuffer( &glVertexBufferResource, 
                                  vertexBufferID, 
                                  cudaGraphicsMapFlagsNone );
}